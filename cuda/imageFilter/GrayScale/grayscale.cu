
#include <hip/hip_runtime.h>
__global__
void cudaGrayScale(float *R, float *G, float *B, float* gray, int n){
    int i = blockDim.x*blockIdx.x + threadIdx.x;
    if(i < n) {
        gray[i] = static_cast<float>((R[i] * 0.21 + G[i] * 0.71 + B[i] * 0.07) / 350.0);
        //gray[i] = static_cast<float>((R[i] + G[i] + B[i]) / (3 * 500.0));
    }
}

void grayscale(float* R, float* G, float* B, float* grayscale, int n){
    int size = n * sizeof(float);
    float *d_R, *d_G, *d_B, *d_gray;
    hipMalloc((void **) &d_R, size);
    hipMemcpy(d_R, R, size, hipMemcpyHostToDevice);
    hipMalloc((void **) &d_G, size);
    hipMemcpy(d_G, G, size, hipMemcpyHostToDevice);
    hipMalloc((void **) &d_B, size);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
    hipMalloc((void **) &d_gray, size);

    cudaGrayScale<<<ceil(n/1024.0), 1024>>>(d_R, d_G, d_B, d_gray, n);
    hipMemcpy(grayscale, d_gray, size, hipMemcpyDeviceToHost);

    hipFree(d_R);
    hipFree(d_G);
    hipFree(d_B);
    hipFree(d_gray);
}
