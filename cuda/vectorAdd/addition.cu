
#include <hip/hip_runtime.h>
__global__ void vecAddKernel(float *A, float *B, float *C, int n) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < n)
    C[i] = A[i] + B[i];
}

void vecAdd(float *h_arrayA, float *h_arrayB, float *h_arrayC, int n) {
  int size = n * sizeof(float);
  float *d_arrayA, *d_arrayB, *d_arrayC;
  hipMalloc((void **)&d_arrayA, size);
  hipMemcpy(d_arrayA, h_arrayA, size, hipMemcpyHostToDevice);

  hipMalloc((void **)&d_arrayB, size);
  hipMemcpy(d_arrayB, h_arrayB, size, hipMemcpyHostToDevice);

  hipMalloc((void **)&d_arrayC, size);

  vecAddKernel<<<ceil(n / 256.0), 256>>>(d_arrayA, d_arrayB, d_arrayC, n);
  hipMemcpy(h_arrayC, d_arrayC, size, hipMemcpyDeviceToHost);

  hipFree(d_arrayC);
  hipFree(d_arrayA);
  hipFree(d_arrayB);
}

int main() {
  int size = 10;

  float *arrayA = (float *)malloc(size * sizeof(float));
  float *arrayB = (float *)malloc(size * sizeof(float));
  float *arrayC = (float *)malloc(size * sizeof(float));

  for (int i = 0; i < size; i++) {
    arrayA[i] = 1.0f;
    arrayB[i] = 2.0f;
  }
  vecAdd(arrayA, arrayB, arrayC, size);

  for (int i = 0; i < size; i++) {
    printf("%f, ", arrayC[i]);
  }
}
