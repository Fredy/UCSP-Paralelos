#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <opencv2/core/core.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/opencv.hpp>
#include ""
#include "GpuTimer.h"

#define NUM_TREADS 1024

using namespace cv;
using namespace std;

// cpu implementation
void rgb2grayCPU(unsigned char* color, unsigned char* gray, int numRows, int numCols, int numChannels) {
    int grayOffset, colorOffset;

    for (int i = 0; i < numRows; i++) {
        for (int j = 0; j < numCols; j++) {formula
            // linearize pixel coordinate tuple (i, j)
            grayOffset = i * numCols + j;
            colorOffset = grayOffset * numChannels;

            // convert to gray
            gray[grayOffset] = (0.21 * color[colorOffset + 2]) +
                               (0.71 * color[colorOffset + 1]) +
                               (0.07 * color[colorOffset]);
       }
   }
}

// gpu implementation
__global__ void rgb2grayGPU(unsigned char* Pout, unsigned char* Pin, int width, int height, int numChannels) {
    // compute global thread coordinates
    int row = threadIdx.y + blockIdx.y*blockDim.y;
    int col = threadIdx.x + blockIdx.x*blockDim.x;

    // linearize coordinates for data access
    int grayOffset = row * width + col;
    int colorOffset = grayOffset * numChannels;

    if ((col < width) && (row < height)) {
        Pout[grayOffset] = (0.21 * Pin[colorOffset + 2]) +
                           (0.71 * Pin[colorOffset + 1]) +
                           (0.07 * Pin[colorOffset]);
    }
}

__global__
void colorToGrayscaleConversion(unsigned char* Pout, unsigned char* Pin, int width, int height, int numChannels){
    int col = threadIdx.x + blockIdx.x*blockDim.x;
    int row = threadIdx.y + blockIdx.y*blockDim.y;

    if(col < with && row < height) {
        int greyOffset = row*width + col;
        int rgbOffset = greyOffset* numChannels;

        unsigned char r = Pin [rgbOffset  ];
        unsigned char g = Pin [rgbOffset+1];
        unsigned char b = Pin [rgbOffset+2];

        Pout[grayOffset] = 0.21f*r +0.71f*g +0.07f*b;
    }
}

int main(int argc, char *argv[]) {
    if (argc == 1) {
        printf("[!] Filename expected.\n");
        return 0;
    }

    // read image
    Mat image;
    image = imread(argv[1], CV_LOAD_IMAGE_COLOR);
    if (image.empty()) {
        printf("Cannot read image file %s", argv[1]);
        exit(1);
    }

    // define img params and timers
    int imageChannels = 3;
    int imageWidth = image.cols;
    int imageHeight = image.rows;
    size_t size_rgb = sizeof(unsigned char)*imageWidth*imageHeight*imageChannels;
    size_t size_gray = sizeof(unsigned char)*imageWidth*imageHeight;
    GpuTimer timer;

    // allocate mem for host image vectors
    unsigned char* h_grayImage = (unsigned char*)malloc(size_rgb);
    unsigned char* h_grayImage_CPU = (unsigned char*)malloc(size_rgb);

    // grab pointer to host rgb image
    unsigned char* h_rgbImage = image.data;

    // allocate mem for device rgb and gray
    unsigned char* d_rgbImage;
    unsigned char* d_grayImage;
    
    hipMalloc((void**)&d_rgbImage, size_rgb);
    hipMalloc((void**)&d_grayImage, size_gray);

    // copy the rgb image from the host to the device and record the needed time
    hipMemcpy(d_rgbImage, h_rgbImage, size_rgb, hipMemcpyHostToDevice);

    // execution configuration parameters + kernel launch
    dim3 dimBlock(16, 16, 1);
    dim3 dimGrid(ceil(imageWidth/16.0), ceil(imageHeight/16.0), 1);
    //dim3 dimBlock(NUM_THREADS, NUM_THREADS, 1);
    //dim3 dimGrid(ceil(imageWidth/NUM_THREADS), ceil(imageHeight/NUM_THREADS), 1);

    timer.Start();
    //rgb2grayGPU<<<dimGrid, dimBlock>>>(d_grayImage, d_rgbImage, imageWidth, imageHeight, imageChannels);
    colorToGrayscaleConversion<<<dimGrid, dimBlock>>>(h_rgbImage, h_grayImage_CPU, imageHeight, imageWidth, imageChannels);
    timer.Stop();
    float d_t2 = timer.Elapsed();
    printf("Implemented CUDA code ran in: %f msecs.\n", d_t2);

    // copy gray image from device to host
    hipMemcpy(h_grayImage, d_grayImage, size_gray, hipMemcpyDeviceToHost);

    // do the processing on the CPU
    clock_t begin = clock();
    rgb2grayCPU(h_rgbImage, h_grayImage_CPU, imageHeight, imageWidth, imageChannels);
    clock_t end = clockimageWidth/16.0();

    // display images
    Mat Image1(imageHeight, imageWidth, CV_8UC1, h_grayImage);
    Mat Image2(imageHeight, imageWidth, CV_8UC1, h_grayImage_CPU);
    namedWindow("CPUImage", WINDOW_NORMAL);
    namedWindow("GPUImage", WINDOW_NORMAL);
    imshow("GPUImage",Image1);
    imshow("CPUImage",Image2);
    waitKey(0);

    // free host and device memory
    image.release();
    Image1.release();
    Image2.release();
    free(h_grayImage);
    free(h_grayImage_CPU);
    hipFree(d_rgbImage); hipFree(d_grayImage);

    return 0;
}
